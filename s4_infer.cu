#include "hip/hip_runtime.h"
#include <NvInfer.h>
#include <NvInferRuntime.h>
#include <NvOnnxParser.h>
#include <bits/stdc++.h>
#include <opencv2/opencv.hpp>

using namespace std;
using namespace nvinfer1;

class Logger : public ILogger {
  void log(Severity severity, const char *msg) noexcept override {
    if (severity <= Severity::kWARNING)
      cout << msg << endl;
  }
};

std::vector<uint8_t> load_engine_file(const std::string &file_name) {
  std::ifstream file(file_name, std::ios::binary | std::ios::ate);
  if (!file) {
    throw std::runtime_error("Unable to open file: " + file_name);
  }

  std::streamsize size = file.tellg();
  if (size <= 0) {
    throw std::runtime_error("Invalid file size: " + file_name);
  }

  std::vector<uint8_t> ret(static_cast<size_t>(size));
  file.seekg(0, std::ios::beg);
  if (!file.read(reinterpret_cast<char *>(ret.data()), size)) {
    throw std::runtime_error("Error read file: " + file_name);
  }
  return ret;
}

int softmax(const float (&rst)[10]) {
  float cache = 0;
  int idx = 0;
  for (int i = 0; i < 10; i += 1) {
    if (rst[i] > cache) {
      cache = rst[i];
      idx = i;
    };
  };
  return idx;
}

int main(int argc, char **argv) {
  Logger logger;

  auto runtime = unique_ptr<IRuntime>(createInferRuntime(logger));
  auto plan = load_engine_file("mnist.engine");
  auto engine = shared_ptr<ICudaEngine>(
      runtime->deserializeCudaEngine(plan.data(), plan.size()));
  auto context =
      unique_ptr<IExecutionContext>(engine->createExecutionContext());

  auto idims = engine->getTensorShape("input");
  auto odims = engine->getTensorShape("output");
  Dims4 inputDims = {1, idims.d[1], idims.d[2], idims.d[3]};
  Dims2 outputDims = {1, 10};
  context->setInputShape("input", inputDims);

  void *buffers[2];
  const int inputIndex = 0;
  const int outputIndex = 1;

  hipMalloc(&buffers[inputIndex], 1 * 28 * 28 * sizeof(float));
  hipMalloc(&buffers[outputIndex], 10 * sizeof(float));

  context->setTensorAddress("input", buffers[inputIndex]);
  context->setTensorAddress("output", buffers[outputIndex]);

  hipStream_t stream;
  hipStreamCreate(&stream);

  vector<string> file_names = {"MNIST/images_test/0/mnist_test_3.png",
                               "MNIST/images_test/1/mnist_test_2.png",
                               "MNIST/images_test/2/mnist_test_1.png",
                               "MNIST/images_test/3/mnist_test_18.png",
                               "MNIST/images_test/4/mnist_test_4.png",
                               "MNIST/images_test/5/mnist_test_8.png",
                               "MNIST/images_test/6/mnist_test_11.png",
                               "MNIST/images_test/7/mnist_test_0.png",
                               "MNIST/images_test/8/mnist_test_61.png",
                               "MNIST/images_test/9/mnist_test_7.png"};

  for (auto file_name : file_names) {
    cv::Mat img0;
    img0 = cv::imread(file_name, 0);
    if (img0.empty()) {
      std::cout << "Could not open or find the image" << std::endl;
      return -1;
    }
    cv::Mat img;
    img0.convertTo(img, CV_32F);

    hipMemcpyAsync(buffers[inputIndex], img.data, 1 * 28 * 28 * sizeof(float),
                    hipMemcpyHostToDevice, stream);

    context->enqueueV3(stream);
    hipStreamSynchronize(stream);

    float rst[10];
    hipMemcpyAsync(&rst, buffers[outputIndex], 1 * 10 * sizeof(float),
                    hipMemcpyDeviceToHost, stream);

    cout << file_name << " result: " << softmax(rst) << endl;
  }

  hipStreamDestroy(stream);
  hipFree(buffers[inputIndex]);
  hipFree(buffers[outputIndex]);
}
